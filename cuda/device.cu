#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" 
{
    bool supportf16() 
    {
        int deviceCount = 0;
        hipError_t error = hipGetDeviceCount(&deviceCount);

        if (error != hipSuccess) 
        {
            return false;
        }

        if (deviceCount == 0) 
        {
            return false;
        }

        for (int deviceIndex = 0; deviceIndex < deviceCount; ++deviceIndex) 
        {
            struct hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, deviceIndex);

            // Check if the GPU supports FP16 (Volta or newer)
            if (deviceProp.major >= 7 && deviceProp.minor >= 0) 
            {
                return true;  // FP16 is supported
            }
        }

        return false; 
    }

    bool is_gpu_available() 
    {
        int deviceCount = 0;
        hipError_t error = hipGetDeviceCount(&deviceCount);

        if (error != hipSuccess) 
        {
            return false;
        }

        if (deviceCount == 0) 
        {
            return false;
        }

        return true;
    }

    int cuda_malloc(void** devPtr, size_t size) 
    {
        hipError_t status = hipMalloc(devPtr, size);
        return status;
    }

    int cuda_free(void* devPtr) 
    {
        hipError_t status = hipFree(devPtr);
        return status;        
    }

    int cuda_memcpy(void* dst, const void* src, size_t size, hipMemcpyKind kind)
    {
        hipError_t status = hipMemcpy(dst, src, size, kind);
        return status;
    }



}
